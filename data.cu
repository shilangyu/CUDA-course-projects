#include "hip/hip_runtime.h"
#include "data.hpp"
#include <cstdint>
#include <iostream>
#include <limits>
#include <random>

Data::Data() {
  _data = std::vector<std::vector<std::uint32_t>>(n_vectors);
  // uniform number generator for 32bit numbers
  auto gen = std::bind(
      std::uniform_int_distribution<std::uint32_t>{
          std::numeric_limits<std::uint32_t>::min(),
          std::numeric_limits<std::uint32_t>::max()},
      // fixed seed
      std::mt19937{0});

  for (auto i = 0; i < n_vectors; i++) {
    std::vector<std::uint32_t> bits(n_32bits);

    for (auto j = 0; j < n_32bits; j++) {
      bits[j] = gen();
    }

    _data[i] = bits;
  }
}

auto Data::to_host_data() const -> std::vector<std::bitset<n_bits>> {
  std::vector<std::bitset<n_bits>> result(n_vectors);

  for (auto i = 0; i < n_vectors; i++) {
    for (auto j = 0; j < n_32bits; j++) {
      for (auto bit = 0; bit < 32; bit++) {
        result[i][n_bits - (j * 32 + bit) - 1] =
            (_data[i][j] >> (32 - bit - 1)) & 0b1;
      }
    }
  }

  return result;
}

auto Data::to_device_data() const -> std::uint32_t ** {
  std::uint32_t **data;

  hipMallocManaged(&data, n_vectors * sizeof(std::uint32_t *));
  for (auto i = 0; i < n_vectors; i++) {
    hipMallocManaged(&data[i], n_32bits * sizeof(std::uint32_t));
    for (auto j = 0; j < n_32bits; j++) {
      data[i][j] = _data[i][j];
    }
  }

  return data;
}

auto Data::delete_device_data(std::uint32_t **data) -> void {
  for (auto i = 0; i < n_vectors; i++) {
    hipFree(data[i]);
  }
  hipFree(data);
}
