#include "hip/hip_runtime.h"
#include "data.cuh"
#include <cstdint>
#include <iostream>
#include <limits>
#include <random>
#include <utility>

Data::Data() {
  _data = std::vector<std::vector<std::uint32_t>>(n_vectors);
  // fixed seed
  std::mt19937 gen(0);
  // uniform number generator for 32bit numbers
  std::uniform_int_distribution<std::uint32_t> dist(
      std::numeric_limits<std::uint32_t>::min(),
      std::numeric_limits<std::uint32_t>::max());

  for (auto i = 0; i < n_vectors; i++) {
    std::vector<std::uint32_t> bits(n_32bits);

    for (auto j = 0; j < n_32bits; j++) {
      bits[j] = dist(gen);
    }

    _data[i] = bits;
  }

  // sprinkle some hamming one pairs
  std::uniform_int_distribution<> vec_pos(0, n_vectors);
  std::uniform_int_distribution<> bit_pos(0, n_bits);
  for (auto i = 0; i < 11; i++) {
    auto from = vec_pos(gen);
    auto to   = vec_pos(gen);
    auto bit  = bit_pos(gen);

    _data[to] = _data[from];
    // toggle a single bit
    auto diff = (_data[from][bit / 32] ^ (1 << (bit % 32)));

    _data[to][bit / 32] = diff;
  }
}

auto Data::to_host_data() const -> std::vector<std::bitset<n_bits>> {
  std::vector<std::bitset<n_bits>> result(n_vectors);

  for (auto i = 0; i < n_vectors; i++) {
    for (auto j = 0; j < n_32bits; j++) {
      for (auto bit = 0; bit < 32; bit++) {
        result[i][n_bits - (j * 32 + bit) - 1] =
            (_data[i][j] >> (32 - bit - 1)) & 0b1;
      }
    }
  }

  return result;
}

auto Data::to_device_data() const -> std::uint32_t ** {
  std::uint32_t **data;

  hipMallocManaged(&data, n_vectors * sizeof(std::uint32_t *));
  for (auto i = 0; i < n_vectors; i++) {
    hipMallocManaged(&data[i], n_32bits * sizeof(std::uint32_t));
    for (auto j = 0; j < n_32bits; j++) {
      data[i][j] = _data[i][j];
    }
  }

  return data;
}

auto Data::delete_device_data(std::uint32_t **data) -> void {
  for (auto i = 0; i < n_vectors; i++) {
    hipFree(data[i]);
  }
  hipFree(data);
}
